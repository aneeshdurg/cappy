#include <cstdint>
#include <stdio.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

#define IPV4(a, b, c, d)                                                       \
  ({                                                                           \
    uint32_t ipv4 = a;                                                         \
    ipv4 = ipv4 * 256 + b;                                                     \
    ipv4 = ipv4 * 256 + c;                                                     \
    ipv4 = ipv4 * 256 + d;                                                     \
    ipv4;                                                                      \
  })

__device__ uint32_t access_u32(char *d_pcap, uint64_t offset) {
  // only aligned accesses are allowed, so we need to align offset to a 32b
  // boundry
  auto rem = offset % 4;
  auto start = offset - rem;

  auto first = *(uint32_t *)(d_pcap + start);
  auto last = *(uint32_t *)(d_pcap + start + 4);

  // get the last `rem` bytes from `first` and the first `4 - rem` bytes from
  // last
  first <<= 8 * (4 - rem);
  last >>= 8 * rem;

  return first | last;
}

/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
__global__ void filterpckts(uint64_t *d_offsets, char *d_pcap, uint32_t *output,
                            uint64_t n_pkts) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i > n_pkts) {
    return;
  }

  constexpr size_t pcap_pkt_header = 16;
  constexpr size_t ethernet_header = 14;
  constexpr size_t header_offset = pcap_pkt_header + ethernet_header;

  auto offset = d_offsets[i];
  uint32_t ip_src = access_u32(d_pcap, offset + header_offset + 12);
  // uint32_t ip_dst = *(int32_t *)(d_pcap + offset + header_offset + 16);
  output[i] = 0;
  // if (ip_src == IPV4(192, 168, 68, 110)) {
  if (ip_src == IPV4(21, 98, 0, 0)) {
    output[i] = 1;
  }
}

#define CuAlloc(sz)                                                            \
  ({                                                                           \
    void *tmp = NULL;                                                          \
    auto err = hipMalloc(&tmp, sz);                                           \
    if (tmp == NULL) {                                                         \
      fprintf(stderr, "Failed to allocate device vector!\n");                  \
      exit(EXIT_FAILURE);                                                      \
    }                                                                          \
    tmp;                                                                       \
  })

extern "C" {

// Returns res[i] = pkt i passed filter ? true : false
char *cappy_main(size_t n_pkts, uint64_t *const pkt_offsets, char *const pcap,
                 size_t pcap_size) {
  // Error code to check return values for CUDA calls
  hipError_t err = hipSuccess;

  auto offsets_size = n_pkts * sizeof(uint64_t);
  uint64_t *d_offsets = (uint64_t *)CuAlloc(offsets_size);
  char *d_pcap = (char *)CuAlloc(pcap_size);
  uint32_t *d_output = (uint32_t *)CuAlloc(n_pkts * sizeof(uint32_t));

  // Copy the host input vectors A and B in host memory to the device input
  // vectors in device memory
  printf("Copy input data from the host memory to the CUDA device\n");
  err =
      hipMemcpy(d_offsets, pkt_offsets, offsets_size, hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    fprintf(stderr,
            "Failed to copy vector A from host to device (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  err = hipMemcpy(d_pcap, pcap, pcap_size, hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    fprintf(stderr,
            "Failed to copy vector A from host to device (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // Launch the Vector Add CUDA Kernel
  int threadsPerBlock = 256;
  int blocksPerGrid = (n_pkts + threadsPerBlock - 1) / threadsPerBlock;
  printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid,
         threadsPerBlock);
  filterpckts<<<blocksPerGrid, threadsPerBlock>>>(d_offsets, d_pcap, d_output,
                                                  n_pkts);
  err = hipGetLastError();

  if (err != hipSuccess) {
    fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  char *h_output = (char *)malloc(n_pkts * sizeof(uint32_t));
  if (h_output == NULL) {
    fprintf(stderr, "failed to allocate host output\n");
    exit(EXIT_FAILURE);
  }

  // Copy the device result vector in device memory to the host result vector
  // in host memory.
  printf("Copy output data from the CUDA device to the host memory\n");
  err = hipMemcpy(h_output, d_output, n_pkts * sizeof(uint32_t),
                   hipMemcpyDeviceToHost);
  if (err != hipSuccess) {
    fprintf(stderr,
            "Failed to copy output from device to host (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // Free device global memory
  err = hipFree(d_offsets);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to free device vector offsets (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  err = hipFree(d_pcap);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to free device vector pcaps (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  err = hipFree(d_output);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to free device vector output (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // Reset the device and exit
  // cudaDeviceReset causes the driver to clean up all state. While
  // not mandatory in normal operation, it is good practice.  It is also
  // needed to ensure correct operation when the application is being
  // profiled. Calling cudaDeviceReset causes all profile data to be
  // flushed before the application exits
  err = hipDeviceReset();
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to deinitialize the device! error=%s\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  printf("Done retval=%p\n", h_output);
  return h_output;
}
}
