#include <cstdint>
#include <stdio.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

#define IPV4(a, b, c, d)                                                       \
  ({                                                                           \
    uint32_t ipv4 = a;                                                         \
    ipv4 = ipv4 * 256 + b;                                                     \
    ipv4 = ipv4 * 256 + c;                                                     \
    ipv4 = ipv4 * 256 + d;                                                     \
    ipv4;                                                                      \
  })

/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
__global__ void filterpckts(uint64_t *d_offsets, char *d_pcap, char *output,
                            uint64_t n_pkts) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i > n_pkts) {
    return;
  }

  constexpr size_t pcap_pkt_header = 16;
  constexpr size_t ethernet_header = 14;
  constexpr size_t header_offset = pcap_pkt_header + ethernet_header;

  auto offset = d_offsets[i];
  uint32_t ip_src = *(uint32_t *)(d_pcap + offset + header_offset + 12);
  // uint32_t ip_dst = *(int32_t *)(d_pcap + offset + header_offset + 16);
  output[i] = 0;
  if (ip_src == IPV4(192, 168, 68, 110)) {
    output[i] = 1;
  }
}

#define CuAlloc(sz)                                                            \
  ({                                                                           \
    void *tmp = NULL;                                                          \
    auto err = hipMalloc(&tmp, sz);                                           \
    if (tmp == NULL) {                                                         \
      fprintf(stderr, "Failed to allocate device vector!\n");                  \
      exit(EXIT_FAILURE);                                                      \
    }                                                                          \
    tmp;                                                                       \
  })

extern "C" {

// Returns res[i] = pkt i passed filter ? true : false
char *cappy_main(size_t n_pkts, uint64_t *const pkt_offsets, char *const pcap,
                 size_t pcap_size) {
  // Error code to check return values for CUDA calls
  hipError_t err = hipSuccess;

  auto offsets_size = n_pkts * sizeof(uint64_t);
  uint64_t *d_offsets = (uint64_t *)CuAlloc(offsets_size);
  char *d_pcap = (char *)CuAlloc(pcap_size);
  char *d_output = (char *)CuAlloc(n_pkts);

  // Copy the host input vectors A and B in host memory to the device input
  // vectors in device memory
  printf("Copy input data from the host memory to the CUDA device\n");
  err =
      hipMemcpy(d_offsets, pkt_offsets, offsets_size, hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    fprintf(stderr,
            "Failed to copy vector A from host to device (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  err = hipMemcpy(d_pcap, pcap, pcap_size, hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    fprintf(stderr,
            "Failed to copy vector A from host to device (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // Launch the Vector Add CUDA Kernel
  int threadsPerBlock = 256;
  int blocksPerGrid = (n_pkts + threadsPerBlock - 1) / threadsPerBlock;
  printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid,
         threadsPerBlock);
  filterpckts<<<blocksPerGrid, threadsPerBlock>>>(d_offsets, d_pcap, d_output,
                                                  n_pkts);
  err = hipGetLastError();

  if (err != hipSuccess) {
    fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  char *h_output = (char *)malloc(n_pkts);
  if (h_output == NULL) {
    fprintf(stderr, "failed to allocate host output\n");
    exit(EXIT_FAILURE);
  }

  // Copy the device result vector in device memory to the host result vector
  // in host memory.
  printf("Copy output data from the CUDA device to the host memory\n");
  err = hipMemcpy(h_output, d_output, n_pkts, hipMemcpyDeviceToHost);
  if (err != hipSuccess) {
    fprintf(stderr,
            "Failed to copy output from device to host (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // Free device global memory
  err = hipFree(d_offsets);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to free device vector offsets (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  err = hipFree(d_pcap);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to free device vector pcaps (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  err = hipFree(d_output);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to free device vector output (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // Reset the device and exit
  // cudaDeviceReset causes the driver to clean up all state. While
  // not mandatory in normal operation, it is good practice.  It is also
  // needed to ensure correct operation when the application is being
  // profiled. Calling cudaDeviceReset causes all profile data to be
  // flushed before the application exits
  err = hipDeviceReset();
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to deinitialize the device! error=%s\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  printf("Done retval=%p\n", h_output);
  return h_output;
}
}